#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C"
{
#include "TArm.h"
#include "timer.h"
#include <stdio.h>
}

// TTV
__global__ void tnsTTVKernel( 
    const tnsValue *X_val, const tnsValue *V_val, tnsValue *Y_val, 
    const tnsIndex X_nnz, const tnsIndex Y_nnz, const tnsIndex V_len)
{
    tnsIndex nnz_i = blockDim.x * blockIdx.x + threadIdx.x;
    if(nnz_i < X_nnz){
        atomicAdd(Y_val + nnz_i / V_len, X_val[nnz_i] * V_val[nnz_i % V_len]);
    }
} 



int tnsHipTTVDentsr(tnsDenseTensor *dentsrY, const tnsDenseTensor *dentsrX, const tnsValueVector *vec, tnsIndex copt_mode){
    // Y的模态信息在使用该函数前在外部确定
    if(copt_mode >= dentsrX->nmodes){
        printf("指定维度不存在\n");
        return 1;
    }
    if(dentsrX->ndims[copt_mode] != vec->nlens || dentsrY->ndims[copt_mode] != 1){
        printf("维度不匹配\n");
        return 1;
    }

    tnsValue* X_val;
    hipMalloc(&X_val, dentsrX->nnz * sizeof(tnsValue));
    hipMemcpy(X_val, dentsrX->values.values, dentsrX->nnz * sizeof(tnsValue), hipMemcpyHostToDevice);

    tnsValue* V_val;
    hipMalloc(&V_val, vec->nlens * sizeof(tnsValue));
    hipMemcpy(V_val, vec->values, vec->nlens * sizeof(tnsValue), hipMemcpyHostToDevice);

    tnsValue* Y_val;
    hipMalloc(&Y_val, dentsrY->nnz * sizeof(tnsValue));
    hipMemcpy(Y_val, dentsrY->values.values, dentsrY->nnz * sizeof(tnsValue), hipMemcpyHostToDevice);

    //dim3 dimBlock(16, 16);
    //dim3 dimGrid(32 / dimBlock.x, 32 / dimBlock.y);
    dim3 block(256);
    dim3 grid(1);
    grid.x = (dentsrX->nnz + block.x - 1) / block.x;

    Timer timer;
    timer_start(&timer);
    //hipLaunchKernelGGL(tnsTTVKernel, dimGrid, dimBlock, 0, 0,  
        // X_val, V_val, Y_val, 
        // dentsrX->nnz, dentsrY->nnz, vec->nlens);  
    hipLaunchKernelGGL(tnsTTVKernel, grid, block, 0, 0,  
        X_val, V_val, Y_val, 
        dentsrX->nnz, dentsrY->nnz, vec->nlens);  

    timer_stop(&timer);
    timer_print_sec(&timer, "DCU Dense TTV");

    hipMemcpy(dentsrY->values.values, Y_val, dentsrY->nnz * sizeof(tnsValue), hipMemcpyDeviceToHost);
    
    hipFree(X_val);
    hipFree(V_val);
    hipFree(Y_val);
    
    return 0;
}

int main(int agrc, char *argv[]){

    // 稠密TTV
    tnsIndex copt_mode = 2;

    // 加载张量tensorx
    tnsDenseTensor tensorx;
    FILE *fp_x = fopen(argv[1], "r");
    tnsLoadDenseTensor(&tensorx, fp_x);
    fclose(fp_x);
    printf("load tensorx\n");
    
    // 加载向量vvc
    tnsValueVector vvc;
    FILE *fp_v = fopen(argv[2], "r");
    tnsNewValueVector(&vvc, tensorx.ndims[copt_mode]);
    tnsLoadValueVector(&vvc, fp_v);
    fclose(fp_v);
    printf("load vector\n");

    tnsDenseTensor tensory;
    tnsNewDenseTensor(&tensory, tensorx.nmodes, tensorx.ndims);
    tensory.ndims[copt_mode] = 1;
    tensory.nnz = tensory.nnz / tensorx.ndims[copt_mode];
    printf("nnz = %u\n", tensory.nnz);
    tnsFreeValueVector(&tensory.values);
    tnsNewValueVector(&tensory.values, tensory.nnz);
    printf("create tensory\n");
    
    tnsHipTTVDentsr(&tensory, &tensorx, &vvc, copt_mode);
    printf("finish single thread compution\n");
    
    FILE *fp_wttv = fopen("/public/software/apps/ghfund/ghfund202107013482/example1/result_dcu.log", "w");
    tnsDumpDenseTensor(&tensory, fp_wttv);
    printf("finish output\n");
    
    fclose(fp_wttv);
    tnsFreeDenseTensor(&tensorx);
    tnsFreeDenseTensor(&tensory);
    tnsFreeValueVector(&vvc);

    return 0;
}