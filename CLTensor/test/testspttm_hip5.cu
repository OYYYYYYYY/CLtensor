#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

extern "C"
{
#include "TArm.h"
#include "timer.h"

}

// SpTTM
__global__ void tnsTTMS1Kernel(
    const tnsValue *X_val, const tnsIndex *X_ndims, const tnsIndex *X_inds,
    const tnsIndex X_nnz, const tnsIndex X_nmodes, const tnsIndex step,
    const tnsValue *M_val, const tnsIndex M_nrows, const tnsIndex M_ncols, 
    const tnsIndex stride, tnsValue *V_val, const tnsIndex V_nlens, 
    const tnsIndex copt_mode)
{
    tnsIndex hash_index;
    tnsIndex hash_step;
    tnsIndex indexs = 1;
    tnsValue data_a;
    tnsValue data_b;
    tnsValue temp;
    // tnsIndex count = 0; 
    // tnsIndex k = 0;
    // // 计算中间结果values
    // for(tnsIndex nnz_i = 0; nnz_i < X_nnz; ++nnz_i){
    //     data_a = X_val[nnz_i];
    //     hash_index = 0;
    //     hash_step = 1;
    //     for(int mode_i = X_nmodes - 1; mode_i >= 0; --mode_i){
    //         if(mode_i != copt_mode){
    //             hash_index += X_inds[mode_i + nnz_i * X_nmodes] * hash_step;
    //             hash_step *= X_ndims[mode_i];
    //         }///< if
    //     }///< for mode_
    //     for(tnsIndex col_i = 0; col_i < M_ncols; ++col_i){
    //         data_b = M_val[X_inds[copt_mode + nnz_i * X_nmodes] * M_ncols + col_i];
    //         // test_val[nnz_i * X_nmodes + col_i] = data_b;
    //         indexs = col_i * step + hash_index;
    //         temp = X_val[nnz_i] * M_val[X_inds[copt_mode + nnz_i * X_nmodes] * M_ncols + col_i];
    //         V_val[indexs] += temp;
    //         test_val[k++] = data_a;
    //         test_val[k++] = data_b;
    //         test_val[k++] = temp;
    //         test_val[k++] = V_val[indexs];
    //     }///< for col_i
    // }///< for nnz_i
    // tnsIndex k = 0;
	// tnsIndex temp;
    tnsIndex nnz_i = blockDim.x * blockIdx.x + threadIdx.x;
    if(nnz_i < X_nnz){
        // data_a = X_val[nnz_i];
        hash_index = 0;
        hash_step = 1;
        for(int mode_i = X_nmodes - 1; mode_i >= 0; --mode_i){
            if(mode_i != copt_mode){
                hash_index += X_inds[mode_i + nnz_i * X_nmodes] * hash_step;
                hash_step *= X_ndims[mode_i];
            }///< if
        }///< for mode_
        for(tnsIndex col_i = 0; col_i < M_ncols; ++col_i){
            // data_b = M_val[X_inds[copt_mode + nnz_i * X_nmodes] * M_ncols + col_i];
            indexs = col_i * step + hash_index;
            V_val[indexs] = X_val[nnz_i] * M_val[X_inds[copt_mode + nnz_i * X_nmodes] * M_ncols + col_i];
        }///< for col_i
    }///< if nnz_i

    
}

__global__ void tnsTTMS2Kernel(
    tnsValue *Y_val, const tnsIndex *Y_ndims, tnsIndex *Y_inds,
    tnsIndex Y_nmodes, tnsValue *V_val, const tnsIndex V_nlens, 
    const tnsIndex copt_mode, const tnsIndex stride)
{
    
    tnsIndex hash_index;
    tnsIndex hash_step;
    tnsIndex indexs = 1;

    tnsIndex nnz_j = 0;
    for(tnsIndex nnz_i = 0; nnz_i < V_nlens; ++nnz_i){
        if(V_val[nnz_i] != 0){
            Y_val[nnz_j] = V_val[nnz_i];
            indexs = nnz_i;
            for(int mode_i = Y_nmodes - 1; mode_i >= 0; --mode_i){
                if(mode_i == copt_mode){
                    Y_inds[mode_i + nnz_j * Y_nmodes] = nnz_i / stride;
                }///< if(mode_i == copt_mode)
                else{
                    Y_inds[mode_i + nnz_j * Y_nmodes] = indexs % Y_ndims[mode_i];
                    indexs = indexs / Y_ndims[mode_i];
                }///< else
            }///< for mode_i
            ++nnz_j;
        }///< if(imdvec.values[nnz_i] != 0)
    }///< for nnz_i

} 



int tnsHipTTMSpatsr(tnsSparseTensor *spatsrY, const tnsSparseTensor *spatsrX, const tnsDenseMatrix *denmat, tnsIndex copt_mode){
    // Y的模态信息在使用该函数前在外部确定
    if(spatsrX->ndims[copt_mode] != denmat->nrows || spatsrY->ndims[copt_mode] != denmat->ncols){
        printf("维度不匹配\n");
        return 1;
    }

    // 获取spatsrY的松散上界
    tnsIndex length = 1;
    for(tnsIndex i = 0; i < spatsrY->nmodes; ++i){
        length  = length * spatsrY->ndims[i];
    } 
    // 获取步长strides
    tnsIndex strides = length / spatsrY->ndims[copt_mode];
    
    // 创建向量imdvec存储中间结果
    tnsValueVector imdvec;
    tnsNewValueVector(&imdvec, length);
    tnsConstantValueVector(&imdvec, 0);

    // 创建向量indvecX存储张量X的索引（按每一个非零元的顺序）
    tnsIndexVector indvecX;
    tnsNewIndexVector(&indvecX, spatsrX->nnz * spatsrX->nmodes);
    for(tnsIndex i = 0; i < spatsrX->nmodes; ++i)
        for(tnsIndex j = 0; j < spatsrX->nnz; ++j)
            indvecX.values[j * spatsrX->nmodes + i] = spatsrX->inds[i].values[j];
        
    printf("Stage 1 : ready to move data to DCU\n");

    // 将host端数据复制到device端
    tnsValue* X_val;
    hipMalloc(&X_val, spatsrX->nnz * sizeof(tnsValue));
    hipMemcpy(X_val, spatsrX->values.values, spatsrX->nnz * sizeof(tnsValue), hipMemcpyHostToDevice);

    tnsIndex* X_ndims;
    hipMalloc(&X_ndims, spatsrX->nmodes * sizeof(tnsIndex));
    hipMemcpy(X_ndims, spatsrX->ndims, spatsrX->nmodes * sizeof(tnsIndex), hipMemcpyHostToDevice);
    
    tnsIndex* X_inds;
    hipMalloc(&X_inds, indvecX.nlens * sizeof(tnsIndex));
    hipMemcpy(X_inds, indvecX.values, indvecX.nlens * sizeof(tnsIndex), hipMemcpyHostToDevice);

    tnsValue* M_val;
    hipMalloc(&M_val, denmat->nrows * denmat->ncols * sizeof(tnsValue));
    hipMemcpy(M_val, denmat->values.values, denmat->nrows * denmat->ncols * sizeof(tnsValue), hipMemcpyHostToDevice);

    tnsValue* V_val;
    hipMalloc(&V_val, imdvec.nlens * sizeof(tnsValue));
    hipMemcpy(V_val, imdvec.values, imdvec.nlens * sizeof(tnsValue), hipMemcpyHostToDevice);

    // 释放向量indvecX
    tnsFreeIndexVector(&indvecX);

    // 启动kernel函数tnsTTMS1Kernel计算中间结果
	printf("Stage 1 : finish move\n");
    
    // dim3 dimBlock(32, 32);
    // dim3 dimGrid(256 / dimBlock.x, 256 / dimBlock.y);
    dim3 block(256);
    dim3 grid(1);
    grid.x = (spatsrX->nnz + block.x - 1) / block.x;

    Timer timer;
    timer_start(&timer);
    // hipLaunchKernelGGL(tnsTTMS1Kernel, dimGrid, dimBlock, 0, 0, 
    //     X_val, X_ndims, X_inds,
    //     spatsrX->nnz, spatsrX->nmodes, strides, 
    //     M_val, denmat->nrows, denmat->ncols, 
    //     denmat->stride, V_val, imdvec.nlens, 
    //     copt_mode);
    hipLaunchKernelGGL(tnsTTMS1Kernel, grid, block, 0, 0, 
        X_val, X_ndims, X_inds,
        spatsrX->nnz, spatsrX->nmodes, strides, 
        M_val, denmat->nrows, denmat->ncols, 
        denmat->stride, V_val, imdvec.nlens, 
        copt_mode);

    timer_stop(&timer);
    // timer_print_sec(&timer, "DCU ttm Stage 1");
    tnsValue time = timer.seconds;
    
    // 将中间结果从device端复制到host端
    hipMemcpy(imdvec.values, V_val, imdvec.nlens * sizeof(tnsValue), hipMemcpyDeviceToHost);

    // 统计中间结果（结果张量）的非零元数
    tnsIndex count = 0;
    for(tnsIndex i = 0; i < imdvec.nlens; ++i)
        if(imdvec.values[i] != 0)
            ++count;
    spatsrY->nnz = count;
    // printf("nnz of spatsrY after computation = %u\n", spatsrY->nnz);
    
    // 释放device端内存
    hipFree(X_val);
    hipFree(X_ndims);
    hipFree(X_inds);
    hipFree(M_val);
    
    //通过准确的非零元数重新分配spatsrY的values和inds在host端的空间
    tnsFreeValueVector(&spatsrY->values);
    tnsNewValueVector(&spatsrY->values, spatsrY->nnz);
    for(tnsIndex mode = 0; mode < spatsrY->nmodes; ++mode){
        tnsFreeIndexVector(&spatsrY->inds[mode]);
        tnsNewIndexVector(&spatsrY->inds[mode], spatsrY->nnz);
    }

    // 创建向量indvecX存储张量X的索引（按每一个非零元的顺序）
    tnsIndexVector indvec;
    tnsNewIndexVector(&indvec, spatsrY->nnz * spatsrY->nmodes);
    for(tnsIndex i = 0; i < spatsrY->nmodes; ++i)
        for(tnsIndex j = 0; j < spatsrY->nnz; ++j)
            indvec.values[j * spatsrY->nmodes + i] = spatsrY->inds[i].values[j];

    printf("Stage 2 : ready to move data to DCU\n");

    // 将host端数据复制到device端
    tnsValue* Y_val;
    hipMalloc(&Y_val, spatsrY->nnz * sizeof(tnsValue));
    hipMemcpy(Y_val, spatsrY->values.values, spatsrY->nnz * sizeof(tnsValue), hipMemcpyHostToDevice);

    tnsIndex* Y_ndims;
    hipMalloc(&Y_ndims, spatsrY->nmodes * sizeof(tnsIndex));
    hipMemcpy(Y_ndims, spatsrY->ndims, spatsrY->nmodes * sizeof(tnsIndex), hipMemcpyHostToDevice);

    tnsIndex* Y_inds;
    hipMalloc(&Y_inds, spatsrY->nnz * spatsrY->nmodes * sizeof(tnsIndex));
    hipMemcpy(Y_inds, indvec.values, indvec.nlens * sizeof(tnsIndex), hipMemcpyHostToDevice);

    hipMemcpy(V_val, imdvec.values, imdvec.nlens * sizeof(tnsValue), hipMemcpyHostToDevice);
    
    // 启动kernel函数tnsTTMS2Kernel将中间结果写回处理到结果张量
    printf("Stage 2 : finish move\n");
    timer_restart(&timer);
    hipLaunchKernelGGL(tnsTTMS2Kernel, 64, 512, 0, 0, 
        Y_val, Y_ndims, Y_inds,
        spatsrY->nmodes, V_val, imdvec.nlens, 
        copt_mode, strides);

    timer_stop(&timer);
    // timer_print_sec(&timer, "DCU ttm Stage 2");
    time += timer.seconds;
    printf("Total time is %.6f\n", time);
    // 创建向量indvecY存储张量spatsrY的索引
    tnsIndexVector indvecY;
    tnsNewIndexVector(&indvecY, spatsrY->nnz * spatsrY->nmodes);

    // 将数据从device端复制到host端
    hipMemcpy(spatsrY->values.values, Y_val, spatsrY->nnz * sizeof(tnsValue), hipMemcpyDeviceToHost);
    hipMemcpy(indvecY.values, Y_inds, spatsrY->nnz * spatsrY->nmodes * sizeof(tnsIndex), hipMemcpyDeviceToHost);

    // 将索引向量indvecY写回到spatsrY的inds中
    for(tnsIndex i = 0; i < spatsrY->nnz; ++i)
        for(tnsIndex j = 0; j < spatsrY->nmodes; ++j)
            spatsrY->inds[j].values[i] = indvecY.values[i * spatsrY->nmodes + j];
    tnsFreeIndexVector(&indvecY);

    hipFree(V_val);
    hipFree(Y_val);
    hipFree(Y_ndims);
    hipFree(Y_inds);
    
    return 0;
}

int main(int agrc, char *argv[]){

    tnsIndex copt_mode = 1;

    tnsSparseTensor tensorx;
    FILE *fp_x = fopen(argv[1], "r");
    tnsLoadSparseTensor(&tensorx, fp_x);
    fclose(fp_x);
    printf("load tensorx\n");

    // 加载稠密矩阵denmat
    tnsDenseMatrix denmat;
    FILE *fp_m = fopen(argv[2], "r");
    tnsLoadDenseMatrix(&denmat, fp_m);
    fclose(fp_m);
    printf("load matrix\n");

    tnsSparseTensor tensory;
    tnsNewSparseTensor(&tensory, tensorx.ndims, tensorx.nmodes, tensorx.nnz);
    tensory.ndims[copt_mode] = denmat.ncols;
    printf("create tensory\n");

    tnsHipTTMSpatsr(&tensory, &tensorx, &denmat, copt_mode);
    printf("finish compution\n");
    
    FILE *fp_w = fopen("/public/software/apps/ghfund/ghfund202107013482/example2/result_dcu5.log", "w");
    tnsDumpSparseTensor(&tensory, fp_w);
    printf("finish output\n");
    fclose(fp_w);
    
    tnsFreeSparseTensor(&tensorx);
    tnsFreeSparseTensor(&tensory);
    tnsFreeDenseMatrix(&denmat);

    return 0;
}